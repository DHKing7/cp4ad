#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>

__global__ void add_kernel( int* C, const int* A, const int* B) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    c[i][j] = a[i][j] + b[i][j]
}

int main(void) {
    struct timeval tv;
    double start_time, end_time;

    gettimeofday(&tv, NULL);
    start_time = (tv.tv_sec) * 1000 + (tv.tv_usec) / 1000;
    const int SIZE = 5;
    const int A[SIZE][SIZE] = {{ 1, 2, 3, 4, 5 }, 
                               { 1, 2, 3, 4, 5 }, 
                               { 1, 2, 3, 4, 5 },
                               { 1, 2, 3, 4, 5 },
                               { 1, 2, 3, 4, 5 }};
    const int B[SIZE][SIZE] = {{ 10, 20, 30, 40, 50 },
                               { 10, 20, 30, 40, 50 },
                               { 10, 20, 30, 40, 50 },
                               { 10, 20, 30, 40, 50 },
                               { 10, 20, 30, 40, 50 }};
    int C[SIZE][SIZE] = { 0, };
    int* dev_a = nullptr;
    int* dev_b = nullptr;
    int* dev_c = nullptr;
    hipMalloc( (void**)&dev_a, S);
    hipMalloc( ());
    hipMalloc(());
    hipMemcpy();
    hipMemcpy();
    add_kernel<<<1, SIZE>>>();

    


    end_time = (tv.tv_sec) * 1000 + (tv.tv_usec) / 1000;
    printf("GPU execution time : %f (sec)", end_time-start_time);
    fflush(stdin);
    return 0;
}